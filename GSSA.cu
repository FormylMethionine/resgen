#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <hiprand/hiprand_mtgp32_host.h>

#define NB 200
#define TPB 256

__global__
void Gillespie(int* X, int nSpecies, double* K, int nReacs, int* M, double
        tstart, double tmax, int N, hiprandStateMtgp32* states) {

    double t;
    double* R = (double*)malloc(nReacs*sizeof(double));
    double Rsum; // sum of reaction rates
    double partialRsum; // partial sum of reaction rates
    int choice; // index of chosen reaction
    double r1, r2; // random numbers
    double tau; // increment of time
    bool exit; // flag to exit the loop
    
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int id=tid; id<N; id += stride) {
        
        t = tstart;

        while (t < tmax) {

            Rsum = 0;

            //Calculte reaction rates
            for (int i=0; i<nReacs; i++) {

                R[i] = K[i];

                for (int j=0; j<nSpecies; j++) {

                    if (M[i*nSpecies + j] < 0) {

                        if (X[j*N + id] >= -M[i*nSpecies + j]) {
                            // Reaction rate can only be non zero if there is
                            // enough reactants to permit reactions
                            // this a safeguard to prevent X from going negative
                            R[i] *= pow((double)X[j*N + id], 
                                        (double)(-M[i*nSpecies + j]));
                        } else R[i] *= 0;
                    }
                }

                Rsum += R[i];
            }

            exit = true;
            for (int i=0; i<nReacs; i++) if (R[i] != 0) exit=false; 
            if (exit) break;

            // Draw two random numbers
            r1 = hiprand_uniform(&states[blockIdx.x]);
            r2 = hiprand_uniform(&states[blockIdx.x]);
            
            // Select reaction to fire
            choice = 0;
            partialRsum = R[choice];
            while (partialRsum < r2*Rsum) {
                choice++;
                partialRsum += R[choice];
            }
            
            // Pass time
            tau = -log(r1)/Rsum;
            t += tau;
            
            // update X
            for (int i=0; i<nSpecies; i++) 
            X[i*N + id] += M[choice*nSpecies + i];

        }

    }

}

int main() {

    int N = 100000;

    int* X;
    double* K;
    int* M;

    hipMallocManaged(&X, 3*N*sizeof(int));
    hipMallocManaged(&K, 4*sizeof(double));
    hipMallocManaged(&M, 3*4*sizeof(int));

    for (int i=0; i<N; i++) X[i] = 10000;
    for (int i=N; i<2*N; i++) X[i] = 0;
    for (int i=2*N; i<3*N; i++) X[i] = 0;

    K[0] = 1;
    K[1] = .002;
    K[2] = .5;
    K[3] = .04;

    M[0] = -1;
    M[1] = 0;
    M[2] = 0;

    M[3] = -2;
    M[4] = 1;
    M[5] = 0;

    M[6] = 2;
    M[7] = -1;
    M[8] = 0;

    M[9] = 0;
    M[10] = -1;
    M[11] = 1;

    hiprandStateMtgp32* states;
    hipMalloc(&states, NB*sizeof(hiprandStateMtgp32));

    mtgp32_kernel_params* kernelParams;
    hipMalloc(&kernelParams, sizeof(kernelParams));
    hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, kernelParams);
    hiprandMakeMTGP32KernelState(states, mtgp32dc_params_fast_11213,
            kernelParams, NB, time(NULL));

    Gillespie <<<NB, TPB>>> (X, 3, K, 4, M, 0.0, 1.0, N, states);
    hipDeviceSynchronize();

    float X_mean[3] = {0, 0, 0};
    for (int i=0; i<3; i++) for (int j=i*N; j<(i+1)*N; j++) X_mean[i] += X[j];
    for (int i=0; i<3; i++) X_mean[i] /= N;
    for (int i=0; i<3; i++) std::cout << X_mean[i] << " ";
    std::cout << std::endl;

    hipFree(&X);
    hipFree(&K);
    hipFree(&M);
    hipFree(&states);
       
}
